#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#define BLOCK_NUM 16
#define THREAD_NUM 64
#define N (BLOCK_NUM * THREAD_NUM)


static void cuda_checker(hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CUDA_CHECK(err) (cuda_checker(err, __FILE__, __LINE__ ))


//
// kernel code
// 

__global__ void add(int *a, int *b, int *c) {

  int tid = blockIdx.x; // handle the data at this index
  
  if(tid < N) {
    c[tid] = a[tid] + b[tid];
  }

}


//
// host code
//

int main(int argc, const char **argv) {



  int a[N], b[N], c[N], d[N], e[N], f[N], g[N];
  int *dev_a, *dev_b, *dev_c, *dev_d, *dev_e, *dev_f, *dev_g;

  for(int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = -i;
    c[i] = -i;
    d[i] = i * i;
    e[i] = i * i;
    f[i] = i * i;
  }

  CUDA_CHECK( hipMalloc((void**)&dev_a, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_b, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_c, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_d, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_e, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_f, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_g, N * sizeof(int)) );

  CUDA_CHECK( hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice) );
  CUDA_CHECK( hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice) );
  CUDA_CHECK( hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice) );
  CUDA_CHECK( hipMemcpy(dev_d, d, N * sizeof(int), hipMemcpyHostToDevice) );
  CUDA_CHECK( hipMemcpy(dev_e, e, N * sizeof(int), hipMemcpyHostToDevice) );
  CUDA_CHECK( hipMemcpy(dev_f, f, N * sizeof(int), hipMemcpyHostToDevice) );


  float time;
  hipEvent_t start, stop;

  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start, 0));


  add<<<N,1>>>(dev_a, dev_b, dev_c, dev_d, dev_e, dev_f);

  CUDA_CHECK( hipMemcpy(g, dev_g, N * sizeof(int), hipMemcpyDeviceToHost) );

  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&time, start, stop));

  printf("Time to generate:  %3.1f ms \n", time);

  // for( int i = 0; i < N; i++ ){
    // printf( "cpu: %d, gpu: %d\n", a[i]+b[i], c[i]);
  // }

  CUDA_CHECK( hipFree(dev_a) );
  CUDA_CHECK( hipFree(dev_b) );
  CUDA_CHECK( hipFree(dev_c) );
  CUDA_CHECK( hipFree(dev_d) );
  CUDA_CHECK( hipFree(dev_e) );
  CUDA_CHECK( hipFree(dev_f) );
  CUDA_CHECK( hipFree(dev_g) );

  hipDeviceReset();

  return 0;
}
