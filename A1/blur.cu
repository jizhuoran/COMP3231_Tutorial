
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

typedef struct {
     unsigned char red,green,blue;
} PPMPixel;

typedef struct {
     int x, y;
     PPMPixel *data;
} PPMImage;

#define CREATOR "COMP3231"
#define RGB_COMPONENT_COLOR 255


static PPMImage *readPPM(const char *filename)
{
         char buff[16];
         PPMImage *img;
         FILE *fp;
         int c, rgb_comp_color;
         //open PPM file for reading
         fp = fopen(filename, "rb");
         if (!fp) {
              fprintf(stderr, "Unable to open file '%s'\n", filename);
              exit(1);
         }

         //read image format
         if (!fgets(buff, sizeof(buff), fp)) {
              perror(filename);
              exit(1);
         }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
         fprintf(stderr, "Invalid image format (must be 'P6')\n");
         exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    //memory allocation for pixel data
    img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    //write the header file
    //image format
    fprintf(fp, "P6\n");

    //comments
    fprintf(fp, "# Created by %s\n",CREATOR);

    //image size
    fprintf(fp, "%d %d\n",img->x,img->y);

    // rgb component depth
    fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(img->data, 3 * img->x, img->y, fp);
    fclose(fp);
}

void changeColorPPM(PPMImage *img)
{
    int i;
    if(img){

         for(i=0;i<img->x*img->y;i++){
              img->data[i].red=RGB_COMPONENT_COLOR-img->data[i].red;
              img->data[i].green=RGB_COMPONENT_COLOR-img->data[i].green;
              img->data[i].blue=RGB_COMPONENT_COLOR-img->data[i].blue;
         }
    }
}

__global__ void my_first_kernel() {
  printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

void your_gaussian_blur_func(PPMImage *img) {
  my_first_kernel<<<4,8>>>();
}

int main(){
    PPMImage *image;
    image = readPPM("input.ppm");

    changeColorPPM(image);
    your_gaussian_blur_func(image);

    writePPM("output.ppm",image);

}