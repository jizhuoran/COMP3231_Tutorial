#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

static void cuda_checker(hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CUDA_CHECK(err) (cuda_checker(err, __FILE__, __LINE__ ))



const int M = 1024, K = 1024, N = 1024;
const int TS = 32;

__global__ void myGEMM1(const float* A,
                        const float* B, 
                            float* C) {
    
    // Thread identifiers
    const int globalRow = threadIdx.x + blockIdx.x * blockDim.x;
    const int globalCol = threadIdx.y + blockIdx.y * blockDim.y;
 
    // Compute a single element (loop over K)
    float acc = 0.0f;
    for (int k=0; k<K; k++) {
        acc += A[k*M + globalRow] * B[globalCol*K + k];
    }
 
    // Store the result
    C[globalCol*M + globalRow] = acc;
}




int main(int argc, const char **argv) {



  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  for(int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  CUDA_CHECK( hipMalloc((void**)&dev_a, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_b, N * sizeof(int)) );
  CUDA_CHECK( hipMalloc((void**)&dev_c, N * sizeof(int)) );

  CUDA_CHECK( hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice) );
  CUDA_CHECK( hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice) );

  add<<<(M / TS,N / TS), (TS, TS)>>>(dev_a, dev_b, dev_c);

  CUDA_CHECK( hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost) );


  for( int i = 0; i < N; i++ ){
    printf( "cpu: %d, gpu: %d\n", a[i]+b[i], c[i]);
  }

  CUDA_CHECK( hipFree(dev_a) );
  CUDA_CHECK( hipFree(dev_b) );
  CUDA_CHECK( hipFree(dev_c) );

  hipDeviceReset();

  return 0;
}