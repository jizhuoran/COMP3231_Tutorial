#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

__global__ void my_first_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x;
}


//
// main code
//

int main(int argc, const char **argv)
{
  float *x;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  checkCudaErrors(hipMallocManaged(&x, nsize*sizeof(float)));

  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>(x);
  getLastCudaError("my_first_kernel execution failed\n");

  // synchronize to wait for kernel to finish, and data copied back

  hipDeviceSynchronize();

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,x[n]);

  // free memory 

  checkCudaErrors(hipFree(x));

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
