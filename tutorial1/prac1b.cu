#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>



//
// kernel code
// 

__global__ void add(int *a, int *b, int *c) {

  int tid = blockIdx.x; // handle the data at this index
  
  if(tid < N) {
    c[tid] = a[tid] + b[tid];
  }

}


//
// host code
//

int main(int argc, const char **argv) {

  int nblocks  = 32, nthreads = 128;

  int N = nblocks * nthreads;

  std::vector<int> a(N), b(N), c(N);
  int *dev_a, *dev_b, *dev_c;

  for(int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  hipMalloc((void**)&dev_a, N * sizeof(int));
  hipMalloc((void**)&dev_b, N * sizeof(int));
  hipMalloc((void**)&dev_c, N * sizeof(int));

  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  add<<<N,1>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);


  for( int i = 0; i < N; i++ ){
    printf( "cpu: %d, gpu: %d\n", a[i]+b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipDeviceReset();

  return 0;
}
