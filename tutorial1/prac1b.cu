#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

__global__ void my_first_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x;
}


//
// main code
//

int main(int argc, const char **argv)
{
  float *h_x, *d_x;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  checkCudaErrors(hipMalloc((void **)&d_x, nsize*sizeof(float)));

  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>(d_x);
  getLastCudaError("my_first_kernel execution failed\n");

  // copy back results and print them out

  checkCudaErrors( hipMemcpy(h_x,d_x,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory 

  checkCudaErrors(hipFree(d_x));
  free(h_x);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
